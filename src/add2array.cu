/*!
 * \file add2array.cu
 *
 * \brief Add two arrays of float.
 *
 * Example code from https://devblogs.nvidia.com/parallelforall/even-easier-introduction-cuda/
 *
 * Compile it with nvcc, the CUDA C++ compiler:
 *
 *    nvcc add2array.cu -o add2array
 *
 * Run with:
 *    ./add2array
 *
 * Profile with:
 *
 *    nvprof ./add2array
 */



#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


/*!
 * \brief Kernel function to add the elements of two arrays.
 *
 * \param[in] n number of elements.
 * \param[in] x array X.
 * \param[in] y array Y
 */
__global__ void
add (int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
    y[i] = x[i] + y[i];
}


int
main (void)
{
  int N = 1 << 20;
  float *x, *y;

  /* Allocate Unified Memory – accessible from CPU or GPU. */
  hipMallocManaged (&x, N * sizeof (float));
  hipMallocManaged (&y, N * sizeof (float));

  /* Initialize x and y arrays on the host. */
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  /* Run kernel on 1M elements on the GPU. */
  add<<<1, 1>>>(N, x, y);

  /* Wait for GPU to finish before accessing on host. */
  hipDeviceSynchronize ();

  /* Check for errors (all values should be 3.0f). */
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax (maxError, fabs (y[i] - 3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  /* Free memory. */
  hipFree (x);
  hipFree (y);
  
  return 0;
}

/* EOF */
